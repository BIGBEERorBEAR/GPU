
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

__global__ void add(float *x, float *y, int N) {
    int thr_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int i = thr_id;

    while (i < N)
    {
        y[i] += x[i];
        i += stride;
    }
}

int main(int argc, char const *argv[])
{
    const int N = argc >= 2 ? std::stoi(argv[1]) : 1e6;
    std::cout << "N = " << N << std::endl;

    float *x, *y;
    float *array_x, *array_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) 
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMalloc(&array_x, N * sizeof(float));
    hipMalloc(&array_y, N * sizeof(float));
    hipMemcpy(array_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(array_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    add<<<512,128>>>(array_x, array_y, N);
    hipMemcpy(y, array_y, N * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(hipDeviceSynchronize());

    free(x);
    free(y);
    hipFree(array_x);
    hipFree(array_y);

    return 0;
}
